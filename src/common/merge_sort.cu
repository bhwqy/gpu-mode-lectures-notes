#include <hip/hip_runtime.h>
#include <cstdio>
#include <algorithm>
#include <random>

void check_cuda_error(hipError_t err) {
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        exit(1);
    }
}

// https://zhuanlan.zhihu.com/p/496102417


int main() {
    return 0;
}

