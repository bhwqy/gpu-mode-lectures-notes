#include <hip/hip_runtime.h>
#include <hipblas.h>

// modified from: https://zhuanlan.zhihu.com/p/657632577

void cublas_sgemm(float *A, float *B, float *C,  size_t M, size_t N, size_t K) {

    hipblasHandle_t handle = nullptr;
    hipblasCreate(&handle);
    hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH);

    static float alpha = 1.0;
    static float beta = 0.0;

    hipblasGemmEx(handle, 
        HIPBLAS_OP_N, 
        HIPBLAS_OP_N, 
        N, M, K, 
        &alpha, 
        B, HIP_R_32F, N, 
        A, HIP_R_32F, K, 
        &beta,  
        C, HIP_R_32F, N, 
        HIPBLAS_COMPUTE_32F,
        HIPBLAS_GEMM_DEFAULT);
}



